#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <stdio.h>
#include <algorithm>
#include <time.h>

using namespace std;

void readInt(int &n, int &m) {
	ifstream fin_n("data/nums.txt");
	fin_n >> n >> m;
}

void readGraph(unsigned long long *neib, int n, int m) {
	ifstream fin_g("data/graph.txt");
	vector<vector<int> > vert;
	vert.resize(n);
	for (int i = 0; i < m; ++i) {
		int u, v;
		fin_g >> u >> v;
		u--, v--;
		neib[i] = ((unsigned long long)u << 32) + v; 
	}
}


__global__ void select_winner_odd(int *parent, unsigned long long *edge_list, int *mark, int *flag, int e_num) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < e_num) {
		unsigned long long temp = edge_list[tid];
		int u, v;
		u = temp & 0xffffffff;
		v = temp >> 32;
		if (parent[u] != parent[v]) {
			parent[max(parent[u], parent[v])] = parent[min(parent[u], parent[v])];
			*flag = 1; 
		} else {
			mark[tid] = 1;
		}  
	}
}

__global__ void select_winner_even(int *parent, unsigned long long *edge_list, int *mark, int *flag, int e_num) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < e_num) {
		unsigned long long temp = edge_list[tid];
		unsigned int u, v;
		u = temp & 0xffffffff;
		v = (temp >> 32) & 0xffffffff;
		if (parent[u] != parent[v]) {
			parent[min(parent[u], parent[v])] = parent[max(parent[u], parent[v])];
			*flag = 1; 
		} else {
			mark[tid] = 1;
		}  
	}
}

__global__ void jump(int *parent, int v_num, int *flag) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < v_num) {	
		int p = parent[tid];
		int p_p = parent[p];
		if (p != p_p) {
			parent[tid] = p_p;
			(*flag) = 1;
		}
	}
}

int main() {	
	int n, m;
	readInt(n, m);
	unsigned long long *h_edge_list, *d_edge_list;
	h_edge_list = (unsigned long long*)malloc(m * sizeof(unsigned long long));
	readGraph(h_edge_list, n, m);
	int h_parent[n], *d_parent;
	int h_mark[m], *d_mark;
	for (int i = 0; i < n; ++i) {
		h_parent[i] = i;
	}
	for (int i = 0; i < m; ++i) {
		h_mark[i] = 0;
	}
	int flag[1], *d_flag;
	int count = 0;
	clock_t beg = clock();
	do {
		flag[0] = 0;
		hipMalloc(&d_parent, n * sizeof(int));
		hipMalloc(&d_edge_list, m * sizeof(unsigned long long));
		hipMalloc(&d_mark, m * sizeof(int));
		hipMalloc(&d_flag, sizeof(int));

		hipMemcpy(d_parent, h_parent, n * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_edge_list, h_edge_list, m * sizeof(unsigned long long), hipMemcpyHostToDevice);
		hipMemcpy(d_mark, h_mark, m * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_flag, flag, sizeof(int), hipMemcpyHostToDevice);

		if (count) {
			select_winner_odd<<<256, 256>>>(d_parent, d_edge_list, d_mark, d_flag, m);
		} else {
			select_winner_even<<<256, 256>>>(d_parent, d_edge_list, d_mark, d_flag, m);
		}
        hipDeviceSynchronize();

		hipMemcpy(flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(h_parent, d_parent, n * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(&d_parent);
		hipFree(&d_edge_list);	
		hipFree(&d_mark);
		hipFree(&d_flag);

		if (!flag[0]) {
			break;
		}
		count ^= 1;
		do {
			flag[0] = 0;
			hipMalloc(&d_flag, sizeof(int));
			hipMalloc(&d_parent, n * sizeof(int));
			hipMemcpy(d_flag, flag, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(d_parent, h_parent, n * sizeof(int), hipMemcpyHostToDevice);
			jump<<<256, 256>>>(d_parent, n, d_flag);
        	hipDeviceSynchronize();	
			hipMemcpy(flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(h_parent, d_parent, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(&d_flag);
			hipFree(&d_parent);
		} while(flag[0]);
	} while(flag);
	cout << float(clock() - beg) / CLOCKS_PER_SEC << endl;
	sort(h_parent, h_parent + n);
	cout << unique(h_parent, h_parent + n) - h_parent;
}
