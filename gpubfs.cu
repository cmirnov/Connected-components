
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdio.h>

using namespace std;

__global__ void bfs(int n, int m, int q_size, int level, int *dist, int *neib, int *off, int *q_size_new, int *q_prev, int *q_next) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < q_size) {
		int u = q_prev[tid];
		for (int i = off[u]; i < (u == n - 1 ? 2 * m : off[u + 1]); ++i) {
			int v = neib[i];
			if (dist[v] == INT_MAX && atomicMin(&dist[v], level + 1) == INT_MAX) {
				int position = atomicAdd(q_size_new, 1);
				q_next[position] = v;
			} 
		}
	}
}

void readInt(int &n, int &m) {
	ifstream fin_n("data/nums.txt");
	fin_n >> n >> m;
}

void readGraph(int *neib, int *off, int n, int m) {
	ifstream fin_g("data/graph.txt");
	vector<vector<int> > vert;
	vert.resize(n);
	for (int i = 0; i < m; ++i) {
		int u, v;
		fin_g >> u >> v;
		u--, v--;
		vert[u].push_back(v);
		vert[v].push_back(u);
	}
	int idx = 0;
	for (int i = 0; i < n; ++i) {
		off[i] = idx;
		for (int j = 0; j < vert[i].size(); ++j) {
			neib[idx] = vert[i][j];
			idx++;
		}
	}

}

int main() {
	int n, m;
	readInt(n, m);
	int *neib, *off;
	neib = (int*)malloc(2 * m * sizeof(int));
	off = (int*)malloc(n * sizeof(int));
	readGraph(neib, off, n, m);
	int dist[n], q_prev[n];
	for (int i = 0; i < n; ++i) {
		dist[i] = INT_MAX;
	}
	int ans = 0;
	int size_new[1] = {0};
	int *d_dist, *d_neib, *d_off, *d_q_prev, *d_q_next, *q_size_new;;
	hipMalloc(&d_dist, n * sizeof(int));
	hipMalloc(&d_neib, 2 * m * sizeof(int));
	hipMalloc(&d_off, n * sizeof(int));
	hipMalloc(&d_q_prev, n * sizeof(int));
	hipMalloc(&d_q_next, n * sizeof(int));
	hipMalloc(&q_size_new, sizeof(int));
	clock_t beg = clock();
	for (int i = 0; i < n; ++i) {
		if (dist[i] == INT_MAX) {
			ans++;
			dist[i] = 0;
			int q_size = 0;
			int level = 0;
			
			for (int j = 0; j < (i == n - 1 ? 2 * m - off[i] : off[i + 1] - off[i]); ++j) {
				q_prev[j] = neib[off[i] + j];
				q_size++;
			} 
			while (q_size > 0) {
				hipMemcpy(q_size_new, size_new, sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(d_dist, dist, n * sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(d_neib, neib, 2 * m * sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(d_off, off, n * sizeof(int), hipMemcpyHostToDevice);
				hipMemcpy(d_q_prev, q_prev, n * sizeof(int), hipMemcpyHostToDevice);
				bfs<<<255, 255>>>(n, m, q_size, level, d_dist, d_neib, d_off, q_size_new, d_q_prev, d_q_next);

				level++;

				hipMemcpy(size_new, q_size_new, sizeof(int), hipMemcpyDeviceToHost);
				q_size = size_new[0];
				size_new[0] = 0;
				hipMemcpy(q_prev, d_q_next, n * sizeof(int), hipMemcpyDeviceToHost);
				hipMemcpy(dist, d_dist, n * sizeof(int), hipMemcpyDeviceToHost);
			}
		}
	}
	cout << float(clock() - beg) / CLOCKS_PER_SEC << endl;
	cout << ans << endl;
	return 0;
}
